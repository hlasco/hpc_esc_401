
#include <hip/hip_runtime.h>
#include <stdio.h> 

#include <sys/time.h>

double getTime(void) {
    struct timeval tv;
    struct timezone tz;
    gettimeofday(&tv, &tz);
    return tv.tv_sec + 1e-6*(double)tv.tv_usec;
}

#define NBIN 1000000000 // Number of bins
#define NUM_BLOCK 500 // Number of thread blocks (max 2^32-1)
#define NUM_THREAD 64 // Number of threads per block (max 1024)



__global__ void cal_pi(
    double *sum, int nbin, double step) {
    int i;
    double x;
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    for (i=idx; i< nbin; i+= blockDim.x*gridDim.x) {
        x = (i+0.5)*step;
        sum[idx] += 4.0/(1.0+x*x);
    }
}




int main(void) {
dim3 dimGrid(NUM_BLOCK,1,1); // Grid dimensions
dim3 dimBlock(NUM_THREAD,1,1); // Block dimensions
double *sumHost, *sumDev; // Pointer to host & device arrays
double pi = 0;
int tid;
double step = 1.0/NBIN; // Step size
size_t size = NUM_BLOCK*NUM_THREAD*sizeof(double);
sumHost = (double *)malloc(size); // array on host
hipMalloc((void **) &sumDev, size);// array on GPU
hipMemset(sumDev, 0, size); // Zero results array

double start, end;
start = getTime();
cal_pi <<<dimGrid, dimBlock>>>(sumDev,NBIN,step); // call CUDA kernel
// Retrieve result from device and store it in host array
hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++)
pi += sumHost[tid];
pi *= step;
// Cleanup
end=getTime();
double delta = end - start;
printf("PI = %.16g computed in %.4g seconds\n", pi, delta);
free(sumHost);
hipFree(sumDev);
}
